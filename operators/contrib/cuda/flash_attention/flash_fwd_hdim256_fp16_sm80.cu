
#include <hip/hip_runtime.h>
// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.
#if USE_FLASH_ATTENTION

#include "flash_fwd_launch_template.h"

namespace flash {

template <>
void run_mha_fwd_<cutlass::half_t, 256>(Flash_fwd_params& params, cudaStream_t stream) {
  run_mha_fwd_hdim256<cutlass::half_t>(params, stream);
}

}  // namespace flash
#endif
